#include "hip/hip_runtime.h"
/*

	Implement your CUDA kernel in this file

*/
#include "cardiacsim_kernels.h"
#include "stdio.h"

__global__ void karrayinit(double **E, double **E_prev, double **R, int n)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y + 1;
	int col = blockIdx.x * blockDim.x + threadIdx.x + 1;
	if (col == 1 && row <= n + 2)
	{
		E[row] = (double *)(E + (n + 2)) + row * (n + 2);
		E_prev[row] = (double *)(E_prev + (n + 2)) + row * (n + 2);
		R[row] = (double *)(R + (n + 2)) + row * (n + 2);
	}
	if (col == 2 && row == 1)
	{
		E[0] = (double *)(E + (n + 2));
		E_prev[0] = (double *)(E_prev + (n + 2));
		R[0] = (double *)(R + (n + 2));
	}
	if (col == 3 && row == 1)
	{
		E[n + 1] = (double *)(E + (n + 2)) + (n + 1) * (n + 2);
		E_prev[n + 1] = (double *)(E_prev + (n + 2)) + (n + 1) * (n + 2);
		R[n + 1] = (double *)(R + (n + 2)) + (n + 1) * (n + 2);
	}
}

__global__ void k1halos(double **E_prev, const int n, const int m)
{

	int row = blockIdx.y * blockDim.y + threadIdx.y + 1;
	int col = blockIdx.x * blockDim.x + threadIdx.x + 1;
	if (col <= n && row <= n)
	{
		if (row == 1)
		{
			E_prev[0][col] = E_prev[2][col];
		}
		else if (row == n)
		{
			E_prev[n + 1][col] = E_prev[n - 1][col];
		}
		if (col == 1)
		{
			E_prev[row][0] = E_prev[row][2];
		}
		else if (col == n)
		{
			E_prev[row][n + 1] = E_prev[row][n - 1];
		}
	}
}

__global__ void k1pde(double **E, double **E_prev, const double alpha, const int n, const int m)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y + 1;
	int col = blockIdx.x * blockDim.x + threadIdx.x + 1;
	if (row <= n && col <= n)
	{
		E[row][col] = E_prev[row][col] + alpha * (E_prev[row][col + 1] + E_prev[row][col - 1] - 4 * E_prev[row][col] + E_prev[row + 1][col] + E_prev[row - 1][col]);
	}
}

__global__ void k1ode(double **E, double **E_prev, double **R,
					  const double alpha, const int n, const int m, const double kk,
					  const double dt, const double a, const double epsilon,
					  const double M1, const double M2, const double b)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y + 1;
	int col = blockIdx.x * blockDim.x + threadIdx.x + 1;
	if (row <= n && col <= n)
	{
		E[row][col] = E[row][col] - dt * (kk * E[row][col] * (E[row][col] - a) * (E[row][col] - 1) + E[row][col] * R[row][col]);
		R[row][col] = R[row][col] + dt * (epsilon + M1 * R[row][col] / (E[row][col] + M2)) * (-R[row][col] - kk * E[row][col] * (E[row][col] - b - 1));
	}
}

__global__ void kswap(double **E, double **E_prev, double **R)
{
	double **tmp = E;
	E = E_prev;
	E_prev = tmp;
}

__global__ void k2(double **E, double **E_prev, double **R, const double alpha, const int n, const int m, const double kk, const double dt, const double a, const double epsilon, const double M1, const double M2, const double b, const int init)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y + 1;
	int col = blockIdx.x * blockDim.x + threadIdx.x + 1;

	if (init == 0 && threadIdx.x == 0 && row <= n)
	{
		E[0] = (double *)(E + n + 2);
		E_prev[0] = (double *)(E_prev + n + 2);
		R[0] = (double *)(R + n + 2);
		E[row] = (double *)(E + n + 2) + row * (n + 2);
		E_prev[row] = (double *)(E_prev + n + 2) + row * (n + 2);
		R[row] = (double *)(R + n + 2) + row * (n + 2);
		E[row - 1] = (double *)(E + n + 2) + (row - 1) * (n + 2);
		E_prev[row - 1] = (double *)(E_prev + n + 2) + (row - 1) * (n + 2);
		R[row - 1] = (double *)(R + n + 2) + (row - 1) * (n + 2);
		E[row + 1] = (double *)(E + n + 2) + (row + 1) * (n + 2);
		E_prev[row + 1] = (double *)(E_prev + n + 2) + (row + 1) * (n + 2);
		R[row + 1] = (double *)(R + n + 2) + (row + 1) * (n + 2);
		E[n + 1] = (double *)(E + n + 2) + (n + 1) * (n + 2);
		E_prev[n + 1] = (double *)(E_prev + n + 2) + (n + 1) * (n + 2);
		R[n + 1] = (double *)(R + n + 2) + (n + 1) * (n + 2);
	} //////////

	__syncthreads();
	if (col <= n && row <= n)
	{
		if (row == 1)
		{
			E_prev[0][col] = E_prev[2][col];
		}
		else if (row == n)
		{
			E_prev[n + 1][col] = E_prev[n - 1][col];
		}
		if (col == 1)
		{
			E_prev[row][0] = E_prev[row][2];
		}
		else if (col == n)
		{
			E_prev[row][n + 1] = E_prev[row][n - 1];
		}
	} ///////////////////////////////////////////
	__syncthreads();
	if (row <= n && col <= n)
	{
		E[row][col] = E_prev[row][col] + alpha * (E_prev[row][col + 1] + E_prev[row][col - 1] - 4 * E_prev[row][col] + E_prev[row + 1][col] + E_prev[row - 1][col]);
		////////////////////////////////////////////
		E[row][col] = E[row][col] - dt * (kk * E[row][col] * (E[row][col] - a) * (E[row][col] - 1) + E[row][col] * R[row][col]);
		R[row][col] = R[row][col] + dt * (epsilon + M1 * R[row][col] / (E[row][col] + M2)) * (-R[row][col] - kk * E[row][col] * (E[row][col] - b - 1));
	}
}

__global__ void k3(double **E, double **E_prev, double **R, const double alpha, const int n, const int m, const double kk, const double dt, const double a, const double epsilon, const double M1, const double M2, const double b, const int init)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y + 1;
	int col = blockIdx.x * blockDim.x + threadIdx.x + 1;

	if (init == 0 && threadIdx.x == 0 && row <= n)
	{
		E[0] = (double *)(E + n + 2);
		E_prev[0] = (double *)(E_prev + n + 2);
		R[0] = (double *)(R + n + 2);
		E[row] = (double *)(E + n + 2) + row * (n + 2);
		E_prev[row] = (double *)(E_prev + n + 2) + row * (n + 2);
		R[row] = (double *)(R + n + 2) + row * (n + 2);
		E[row - 1] = (double *)(E + n + 2) + (row - 1) * (n + 2);
		E_prev[row - 1] = (double *)(E_prev + n + 2) + (row - 1) * (n + 2);
		R[row - 1] = (double *)(R + n + 2) + (row - 1) * (n + 2);
		E[row + 1] = (double *)(E + n + 2) + (row + 1) * (n + 2);
		E_prev[row + 1] = (double *)(E_prev + n + 2) + (row + 1) * (n + 2);
		R[row + 1] = (double *)(R + n + 2) + (row + 1) * (n + 2);
		E[n + 1] = (double *)(E + n + 2) + (n + 1) * (n + 2);
		E_prev[n + 1] = (double *)(E_prev + n + 2) + (n + 1) * (n + 2);
		R[n + 1] = (double *)(R + n + 2) + (n + 1) * (n + 2);
	} //////////

	__syncthreads();
	if (col <= n && row <= n)
	{
		if (row == 1)
		{
			E_prev[0][col] = E_prev[2][col];
		}
		else if (row == n)
		{
			E_prev[n + 1][col] = E_prev[n - 1][col];
		}
		if (col == 1)
		{
			E_prev[row][0] = E_prev[row][2];
		}
		else if (col == n)
		{
			E_prev[row][n + 1] = E_prev[row][n - 1];
		}
	} ///////////////////////////////////////////

	__syncthreads();
	if (row <= n && col <= n)
	{
		double ee = E_prev[row][col] + alpha * (E_prev[row][col + 1] + E_prev[row][col - 1] - 4 * E_prev[row][col] + E_prev[row + 1][col] + E_prev[row - 1][col]);
		double rr = R[row][col];
		double e2 = ee - dt * (kk * ee * (ee - a) * (ee - 1) + ee * rr);
		E[row][col] = e2;
		R[row][col] = rr + dt * (epsilon + M1 * rr / (e2 + M2)) * (-rr - kk * e2 * (e2 - b - 1));
	}
}

__global__ void k4(double **E, double **E_prev, double **R, const double alpha, const int n, const int m, const double kk, const double dt, const double a, const double epsilon, const double M1, const double M2, const double b, const int init, const int bx, const int by)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y + 1;
	int col = blockIdx.x * blockDim.x + threadIdx.x + 1;
	if (init == 0 && threadIdx.x == 0 && row <= n && col <= n)
	{
		E[0] = (double *)(E + n + 2);
		E_prev[0] = (double *)(E_prev + n + 2);
		R[0] = (double *)(R + n + 2);
		E[row] = (double *)(E + n + 2) + row * (n + 2);
		E_prev[row] = (double *)(E_prev + n + 2) + row * (n + 2);
		R[row] = (double *)(R + n + 2) + row * (n + 2);
		E[row - 1] = (double *)(E + n + 2) + (row - 1) * (n + 2);
		E_prev[row - 1] = (double *)(E_prev + n + 2) + (row - 1) * (n + 2);
		R[row - 1] = (double *)(R + n + 2) + (row - 1) * (n + 2);
		E[row + 1] = (double *)(E + n + 2) + (row + 1) * (n + 2);
		E_prev[row + 1] = (double *)(E_prev + n + 2) + (row + 1) * (n + 2);
		R[row + 1] = (double *)(R + n + 2) + (row + 1) * (n + 2);
		E[n + 1] = (double *)(E + n + 2) + (n + 1) * (n + 2);
		E_prev[n + 1] = (double *)(E_prev + n + 2) + (n + 1) * (n + 2);
		R[n + 1] = (double *)(R + n + 2) + (n + 1) * (n + 2);
	} //////////

	extern __shared__ double shared_E_prev[];

	int lrow = threadIdx.y + 1;
	int lcol = threadIdx.x + 1;
	__syncthreads();
	if (col <= n && row <= n)
	{
		shared_E_prev[lrow * (bx + 2) + lcol] = E_prev[row][col];
		if (lrow == 1)
		{
			shared_E_prev[lcol] = row == 1 ? E_prev[2][col] : E_prev[row - 1][col];
		}

		if (lrow == by || row == n)
		{
			shared_E_prev[(lrow + 1) * (bx + 2) + lcol] = row == n ? E_prev[n - 1][col] : E_prev[row + 1][col];
		}

		if (lcol == 1)
		{
			shared_E_prev[lrow * (bx + 2)] = col == 1 ? E_prev[row][2] : E_prev[row][col - 1];
		}

		if (lcol == bx || col == n)
		{
			shared_E_prev[lrow * (bx + 2) + lcol + 1] = col == n ? E_prev[row][n - 1] : E_prev[row][col + 1];
		}

	} ///////////////////////////////////////////
	__syncthreads();
	if (row <= n && col <= n)
	{
		double ee = shared_E_prev[lrow * (bx + 2) + lcol] + alpha * (shared_E_prev[lrow * (bx + 2) + lcol + 1] + shared_E_prev[lrow * (bx + 2) + lcol - 1] - 4 * shared_E_prev[lrow * (bx + 2) + lcol] + shared_E_prev[(lrow + 1) * (bx + 2) + lcol] + shared_E_prev[(lrow - 1) * (bx + 2) + lcol]);
		double rr = R[row][col];
		double e2 = ee - dt * (kk * ee * (ee - a) * (ee - 1) + ee * rr);
		E[row][col] = e2;
		R[row][col] = rr + dt * (epsilon + M1 * rr / (e2 + M2)) * (-rr - kk * e2 * (e2 - b - 1));
	}
}
